// BOARD_SIZE = 8;

int main(int argc, char *argv[]) {
  int possibilities[8] = {1, 1, 1, 1, 1, 1, 1, 1};
  int attacks[12] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};



  dim3 threadsPerBlock(8);
  dim3 numBlocks(1);

  runKernel<<<numBlocks, threadsPerBlock>>>()

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
}
